#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""

#include <GL/glew.h>
#include <GLFW/glfw3.h>

#include <glad/glad.h>

#include <stdlib.h>
#include <stdio.h>
#include <stdbool.h>

#include <cuda_gl_interop.h>

#include "Interop.h"
#include "CudaHelpers.cuh"

#include "imgui/imgui_impl_glfw.h"
#include "imgui/imgui_impl_opengl3.h"

const char* glsl_version;
bool show_demo_window = true;
bool show_another_window = false;
ImVec4 clear_color;

surface<void, cudaSurfaceType2D> surf;

union pxl_rgbx_24
{
    uint1       b32;

    struct {
        unsigned  r : 8;
        unsigned  g : 8;
        unsigned  b : 8;
        unsigned  na : 8;
    };
};

__device__ float remap(float value, float min, float max)
{
    return (value - min) / (max - min);
}

__global__ void test_kernel(int w, int h)
{
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    const int idx = (blockDim.x * blockIdx.x) + threadIdx.x;
    const int x = idx % w;
    const int y = idx / w;
    
    union pxl_rgbx_24 rgbx;

    rgbx.r = (int)255 * remap(x, 0.f, (float)w);
    rgbx.g = (int)255 * remap(y, 0.f, (float)h);
    rgbx.b = 0;
    rgbx.na = 255;

    surf2Dwrite(
        rgbx.b32,
        surf,
        x * sizeof(rgbx),
        y,
        hipBoundaryModeZero
    );
}

static void glfw_error_callback(int error, const char* description)
{
    fputs(description, stderr);
}

static void glfw_window_size_callback(GLFWwindow* window, int width, int height)
{
    // get context
    Interop* interop = (Interop*)glfwGetWindowUserPointer(window);
    interop->set_size(width, height);
}

static void glfw_init(GLFWwindow** window, const int width, const int height)
{
    //
    // INITIALIZE GLFW/GLAD
    //

    glfwSetErrorCallback(glfw_error_callback);
    //glfwSetKeyCallback(*window, glfw_key_callback);
    glfwSetFramebufferSizeCallback(*window, glfw_window_size_callback);
    //glfwSetCursorPosCallback(*window, glfw_mouse_callback);
    //glfwSetMouseButtonCallback(*window, glfw_mouse_button_callback);

    if (!glfwInit())
        exit(EXIT_FAILURE);

    glsl_version = "#version 330";

    glfwWindowHint(GLFW_DEPTH_BITS, 0);
    glfwWindowHint(GLFW_STENCIL_BITS, 0);

    glfwWindowHint(GLFW_SRGB_CAPABLE, GL_TRUE);

    glfwWindowHint(GLFW_CONTEXT_VERSION_MAJOR, 4);
    glfwWindowHint(GLFW_CONTEXT_VERSION_MINOR, 5);

    glfwWindowHint(GLFW_OPENGL_PROFILE, GLFW_OPENGL_CORE_PROFILE);

#ifdef PXL_FULLSCREEN
    GLFWmonitor* monitor = glfwGetPrimaryMonitor();
    const GLFWvidmode* mode = glfwGetVideoMode(monitor);
    *window = glfwCreateWindow(mode->width, mode->height, "GLFW / CUDA Interop", monitor, NULL);
#else
    *window = glfwCreateWindow(width, height, "GLFW / CUDA Interop", NULL, NULL);
#endif

    if (*window == NULL)
    {
        glfwTerminate();
        exit(EXIT_FAILURE);
    }

    glfwMakeContextCurrent(*window);

    if (glewInit() != GLEW_OK)
        exit(EXIT_FAILURE);

    // set up GLAD
    gladLoadGLLoader((GLADloadproc)glfwGetProcAddress);

    // ignore vsync for now
    glfwSwapInterval(0);

    // only copy r/g/b
    glColorMask(GL_TRUE, GL_TRUE, GL_TRUE, GL_FALSE);
}

static void imgui_init(GLFWwindow** window) {
    // Setup Dear ImGui context
    IMGUI_CHECKVERSION();
    ImGui::CreateContext();
    ImGuiIO& io = ImGui::GetIO(); (void)io;
    //io.ConfigFlags |= ImGuiConfigFlags_NavEnableKeyboard;     // Enable Keyboard Controls
    //io.ConfigFlags |= ImGuiConfigFlags_NavEnableGamepad;      // Enable Gamepad Controls

    // Setup Dear ImGui style
    //ImGui::StyleColorsDark();
    ImGui::StyleColorsClassic();

    // Setup Platform/Renderer backends
    ImGui_ImplGlfw_InitForOpenGL(*window, true);
    ImGui_ImplOpenGL3_Init(glsl_version);

    // Our state
    show_demo_window = true;
    show_another_window = false;
    ImVec4 clear_color;
}

hipError_t kernel_launcher(
    hipArray_const_t array,
    const int         width,
    const int         height,
    hipEvent_t       event,
    hipStream_t      stream)
{
    hipError_t cuda_err;

    // cuda_err = hipEventRecord(event,stream);

    cuda_err = cudaBindSurfaceToArray(surf, array);

    if (cuda_err)
        return cuda_err;

    const int blocks = (width * height + 256 - 1) / 256;

    // cuda_err = hipEventRecord(event,stream);

    if (blocks > 0)
        test_kernel <<< blocks, 256, 0, stream >>> (width, height);

    // cuda_err = hipStreamWaitEvent(stream,event,0);

    return hipSuccess;
}

int main()
{
    int width = 1024;
    int height = 1024;

    GLFWwindow* window;
    glfw_init(&window, width, height);
    imgui_init(&window);

    //
    // CREATE CUDA STREAM & EVENT
    //
    hipStream_t stream;
    hipEvent_t  event;

    checkCudaErrors(hipStreamCreateWithFlags(&stream, hipStreamDefault));   // optionally ignore default stream behavior
    checkCudaErrors(hipEventCreateWithFlags(&event, hipEventBlockingSync)); // | hipEventDisableTiming);

    //
    // CREATE INTEROP
    //
    // TESTING -- DO NOT SET TO FALSE, ONLY TRUE IS RELIABLE
    Interop* interop = new Interop(true, 2);

    checkCudaErrors(interop->set_size(width, height));

    //
    // SET USER POINTER AND CALLBACKS
    //
    glfwSetWindowUserPointer(window, interop);
    glfwSetFramebufferSizeCallback(window, glfw_window_size_callback);

    while (!glfwWindowShouldClose(window))
    {
        //
        // EXECUTE CUDA KERNEL ON RENDER BUFFER
        //
        int width, height;
        hipArray_t cuda_array;

        interop->get_size(&width, &height);
        checkCudaErrors(interop->map(stream));
        checkCudaErrors(kernel_launcher(interop->array_get(),width,height,event,stream));
        checkCudaErrors(interop->unmap(stream));

        //
        // BLIT & SWAP FBO
        // 
        interop->blit();
        //interop->clear();
        interop->swap();

        // Start the Dear ImGui frame
        ImGui_ImplOpenGL3_NewFrame();
        ImGui_ImplGlfw_NewFrame();
        ImGui::NewFrame();

        // 1. Show the big demo window (Most of the sample code is in ImGui::ShowDemoWindow()! You can browse its code to learn more about Dear ImGui!).
        if (show_demo_window)
            ImGui::ShowDemoWindow(&show_demo_window);

        // 2. Show a simple window that we create ourselves. We use a Begin/End pair to created a named window.
        {
            static float f = 0.0f;
            static int counter = 0;

            ImGui::Begin("Hello, world!");                          // Create a window called "Hello, world!" and append into it.

            ImGui::Text("This is some useful text.");               // Display some text (you can use a format strings too)
            ImGui::Checkbox("Demo Window", &show_demo_window);      // Edit bools storing our window open/close state
            ImGui::Checkbox("Another Window", &show_another_window);

            ImGui::SliderFloat("float", &f, 0.0f, 1.0f);            // Edit 1 float using a slider from 0.0f to 1.0f
            ImGui::ColorEdit3("clear color", (float*)&clear_color); // Edit 3 floats representing a color

            if (ImGui::Button("Button"))                            // Buttons return true when clicked (most widgets return true when edited/activated)
                counter++;
            ImGui::SameLine();
            ImGui::Text("counter = %d", counter);

            ImGui::Text("Application average %.3f ms/frame (%.1f FPS)", 1000.0f / ImGui::GetIO().Framerate, ImGui::GetIO().Framerate);
            ImGui::End();
        }

        // 3. Show another simple window.
        if (show_another_window)
        {
            ImGui::Begin("Another Window", &show_another_window);   // Pass a pointer to our bool variable (the window will have a closing button that will clear the bool when clicked)
            ImGui::Text("Hello from another window!");
            if (ImGui::Button("Close Me"))
                show_another_window = false;
            ImGui::End();
        }

        // Rendering
        ImGui::Render();
        ImGui_ImplOpenGL3_RenderDrawData(ImGui::GetDrawData());

        glfwSwapBuffers(window); 
        glfwPollEvents();
    }

    //
    // CLEANUP
    //
    delete interop;
    glfwDestroyWindow(window);
    glfwTerminate();

    hipDeviceReset();
    exit(EXIT_SUCCESS);
}
